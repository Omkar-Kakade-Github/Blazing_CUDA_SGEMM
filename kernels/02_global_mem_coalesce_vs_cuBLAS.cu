#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <ctime>
#include <cmath>

#define MATRIX_SIZE 4096
#define BLOCKSIZE 32

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void sgemm_global_mem_coalesce(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C) {
    const int cRow = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const int cCol = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

    if (cRow < M && cCol < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; i++) {
            sum += A[cRow * K + i] * B[i * N + cCol];
        }

        C[cRow * N + cCol] = alpha * sum + beta * C[cRow * N + cCol];
    }
}

// Function to fill a matrix with random values
void fillMatrixRandom(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = (static_cast<float>(std::rand())) / RAND_MAX; // Random float in [0, 1)
    }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {
    // Seed the random number generator
    std::srand(static_cast<unsigned int>(std::time(nullptr)));

    // Matrix dimensions
    const int M = MATRIX_SIZE;
    const int N = MATRIX_SIZE;
    const int K = MATRIX_SIZE;

    // Host memory allocation
    float* h_A = (float*)malloc(M * K * sizeof(float));
    float* h_B = (float*)malloc(K * N * sizeof(float));
    float* h_C_coalesce = (float*)malloc(M * N * sizeof(float));
    float* h_C_cublas = (float*)malloc(M * N * sizeof(float));

    // Fill matrices A and B with random values
    fillMatrixRandom(h_A, M, K);
    fillMatrixRandom(h_B, K, N);

    // Initialize result matrices to zero
    for (int i = 0; i < M * N; ++i) {
        h_C_coalesce[i] = 0.0f;
        h_C_cublas[i] = 0.0f;
    }

    // CUDA setup
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void **)&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **)&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **)&d_C, M * N * sizeof(float)));

    // Copy matrices A and B to the device
    CHECK_CUDA(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // --- Global Memory Coalescing SGEMM ---
    dim3 blockDim(32 * 32);
    dim3 gridDim((M + BLOCKSIZE - 1) / BLOCKSIZE, (N + BLOCKSIZE - 1) / BLOCKSIZE);

    double start_time = get_time();
    sgemm_global_mem_coalesce<<<gridDim, blockDim>>>(M, N, K, 1.0f, d_A, d_B, 0.0f, d_C);
    CHECK_CUDA(hipDeviceSynchronize());
    double end_time = get_time();

    CHECK_CUDA(hipMemcpy(h_C_coalesce, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // --- cuBLAS SGEMM ---
    double start_cublas = get_time();
    float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
    CHECK_CUDA(hipDeviceSynchronize());
    double end_cublas = get_time();

    CHECK_CUDA(hipMemcpy(h_C_cublas, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    double gflops = 2.0 * M * N * K * 1e-9; // Total operations in GFLOPs

    // Calculate performance relative to cuBLAS
    double coalesce_time = end_time - start_time;
    double cublas_time = end_cublas - start_cublas;
    double coalesce_gflops = gflops / coalesce_time;
    double cublas_gflops = gflops / cublas_time;
    double relative_performance = (cublas_time / coalesce_time) * 100;

    // Print results
    printf("Global Memory Coalescing SGEMM Time: %f seconds\n", coalesce_time);
    printf("cuBLAS SGEMM Time: %f seconds\n", cublas_time);
    printf("Global Memory Coalescing SGEMM GFLOPs: %f\n", coalesce_gflops);
    printf("cuBLAS SGEMM GFLOPs: %f\n", cublas_gflops);
    printf("Performance relative to cuBLAS: %f%%\n", relative_performance);

    // Clean up
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(handle));

    free(h_A);
    free(h_B);
    free(h_C_coalesce);
    free(h_C_cublas);

    return 0;
}
