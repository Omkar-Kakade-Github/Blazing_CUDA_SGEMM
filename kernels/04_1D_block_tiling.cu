#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <ctime>

#define MATRIX_SIZE 4096
#define BLOCK_SIZE 256 
#define BM 64
#define BN 64
#define BK 8
#define TM 4

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

// Error checking macros for CUDA and cuBLAS
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}


// so grid.x covers columns and grid.y covers rows.
__global__ void sgemm1DBlockTiling(int M, int N, int K, float alpha,
                                   const float *A, const float *B, float beta,
                                   float *C) {
    // Determine which tile of C this block is responsible for.
    const unsigned int tileRow = blockIdx.y; // each tile covers BM rows
    const unsigned int tileCol = blockIdx.x; // each tile covers BN columns

    // We assume BN=64 threads per row, so we have 256/64 = 4 thread rows.
    const int threadCol = threadIdx.x % BN;      // 0..63
    const int threadRow = threadIdx.x / BN;        // 0..3

    // Allocate shared memory tiles.
    __shared__ float As[BM * BK]; // tile from A: 64x8 elements
    __shared__ float Bs[BK * BN]; // tile from B: 8x64 elements

    // Adjust pointers to the beginning of the tile in global memory.
    // Each tile of A starts at row (tileRow*BM) and B starts at column (tileCol*BN).
    A += tileRow * BM * K;
    B += tileCol * BN;
    C += tileRow * BM * N + tileCol * BN;

    // Each thread maintains an accumulator for TM output rows.
    float threadResults[TM];
    #pragma unroll
    for (int i = 0; i < TM; ++i)
        threadResults[i] = 0.0f;

    // Loop over the K dimension in blocks of BK.
    for (int bkIdx = 0; bkIdx < K; bkIdx += BK) {
        // Load A tile (size BM x BK) into shared memory.
        // Use a strided loop so that all 256 threads cooperate.
        for (int idx = threadIdx.x; idx < BM * BK; idx += blockDim.x) {
            int row = idx / BK;
            int col = idx % BK;
            As[idx] = A[row * K + col];
        }
        // Load B tile (size BK x BN) into shared memory.
        for (int idx = threadIdx.x; idx < BK * BN; idx += blockDim.x) {
            int row = idx / BN;
            int col = idx % BN;
            Bs[idx] = B[row * N + col];
        }
        __syncthreads();

        // Compute dot products for each element of the output sub-tile.
        // For each position in the K dimension tile.
        for (int dotIdx = 0; dotIdx < BK; ++dotIdx) {
            float tmpB = Bs[dotIdx * BN + threadCol]; // value from B tile
            // Each thread computes TM consecutive rows.
            for (int resIdx = 0; resIdx < TM; ++resIdx) {
                // (threadRow*TM + resIdx) gives the row in the tile
                threadResults[resIdx] += 
                    As[(threadRow * TM + resIdx) * BK + dotIdx] * tmpB;
            }
        }
        __syncthreads();

        // Advance A and B pointers to the next tile in K.
        A += BK;
        B += BK * N;
    }

    // Write the computed results from registers back to global memory.
    for (int resIdx = 0; resIdx < TM; ++resIdx) {
        // Row index in the output tile: (threadRow * TM + resIdx)
        int row = threadRow * TM + resIdx;
        C[row * N + threadCol] = alpha * threadResults[resIdx] + 
                                 beta * C[row * N + threadCol];
    }
}

// Function to fill a matrix with random values
void fillMatrixRandom(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = static_cast<float>(std::rand()) / RAND_MAX; // Random float in [0, 1)
    }
}

// Function to get the current time in seconds
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {
    // Seed the random number generator
    std::srand(static_cast<unsigned int>(std::time(nullptr)));

    // Matrix dimensions
    const int M = MATRIX_SIZE;
    const int N = MATRIX_SIZE;
    const int K = MATRIX_SIZE;

    // Host memory allocation
    float* h_A = (float*)malloc(M * K * sizeof(float));
    float* h_B = (float*)malloc(K * N * sizeof(float));
    float* h_C_1D = (float*)malloc(M * N * sizeof(float));
    float* h_C_cublas = (float*)malloc(M * N * sizeof(float));

    // Fill matrices A and B with random values
    fillMatrixRandom(h_A, M, K);
    fillMatrixRandom(h_B, K, N);

    // Initialize result matrices to zero
    for (int i = 0; i < M * N; ++i) {
        h_C_1D[i] = 0.0f;
        h_C_cublas[i] = 0.0f;
    }

    // CUDA setup
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void **)&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **)&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **)&d_C, M * N * sizeof(float)));

    // Copy matrices A and B to the device
    CHECK_CUDA(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // --- 1D Block Tiling SGEMM ---
    dim3 blockDim(BLOCK_SIZE); // Threads per block
    dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM)); // Blocks in grid

    double start_time = get_time();
    sgemm1DBlockTiling<<<gridDim, blockDim>>>(M, N, K, 1.0f, d_A, d_B, 0.0f, d_C);
    CHECK_CUDA(hipDeviceSynchronize());
    double end_time = get_time();

    CHECK_CUDA(hipMemcpy(h_C_1D, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // --- cuBLAS SGEMM ---
    double start_cublas = get_time();
    float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
    CHECK_CUDA(hipDeviceSynchronize());
    double end_cublas = get_time();

    CHECK_CUDA(hipMemcpy(h_C_cublas, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Calculate performance metrics
    double gflops = 2.0 * M * N * K * 1e-9; // Total operations in GFLOPs

    double one_D_tile_time = end_time - start_time;
    double cublas_time = end_cublas - start_cublas;
    double one_D_tile = gflops / one_D_tile_time;
    double cublas_gflops = gflops / cublas_time;
    double relative_performance = (cublas_time / one_D_tile_time) * 100;

    // Print results
    printf("1D Block Tiling SGEMM Time: %f seconds\n", one_D_tile_time);
    printf("cuBLAS SGEMM Time: %f seconds\n", cublas_time);
    printf("1D Block Tiling SGEMM GFLOPs: %f\n", one_D_tile);
    printf("cuBLAS SGEMM GFLOPs: %f\n", cublas_gflops);
    printf("Performance relative to cuBLAS: %f%%\n", relative_performance);

    // Clean up
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasDestroy(handle));

    free(h_A);
    free(h_B);
    free(h_C_1D);
    free(h_C_cublas);

    return 0;
}
