#include "sgemm_kernels.h"
#include <cstdio>

void run_cublas_sgemm(hipblasHandle_t handle, int M, int N, int K, 
                      float alpha, float* d_A, float* d_B, 
                      float beta, float* d_C) {

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, HIP_R_32F,
               N, d_A, HIP_R_32F, K, &beta, d_C, HIP_R_32F, N,
               HIPBLAS_COMPUTE_32F_FAST_TF32, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}
